#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include "math.h"
#include "stdlib.h"

__global__ void addKernel(char* arr, double LEFT, double BOTTOM, double RIGHT, double TOP, double SIZE, double MaxIters)
{
    int ind = blockIdx.x * blockDim.x + threadIdx.x;
    if (ind < (int)SIZE * SIZE)
    {
        int x, y, count;
        double zr, zi, betar, betai, a, b;
        double rsquared, isquared, mr, mi, msquared;

        y = ind / SIZE;
        x = ind - SIZE * y;

        zr = 1.0;
        zi = 0.0;
        betar = LEFT + x * (RIGHT - LEFT) / SIZE;
        betai = BOTTOM + y * (TOP - BOTTOM) / SIZE;

        a = 0.5 * (1 - betar);
        b = -0.5 * betai;
        rsquared = zr * zr;
        isquared = zi * zi;

        for (count = 0; rsquared + isquared >= 2.25 / (betar * betar + betai * betai) && count < MaxIters; count++)
        {
            mr = 2 * a - 1 + exp(a * zr - b * zi) * cos(a * zi + b * zr);
            mi = 2 * b + exp(a * zr - b * zi) * sin(a * zi + b * zr);
            msquared = mr * mr + mi * mi;
            zr = 1 - 2 * (a * mr + b * mi) / msquared;
            zi = 2 * (a * mi - b * mr) / msquared;
            rsquared = zr * zr;
            isquared = zi * zi;
        }

        if (rsquared + isquared >= 2.25 / (betar * betar + betai * betai))
            arr[x * int(SIZE) + y] = '*';
        else
            arr[x * int(SIZE) + y] = '.';
    }
}

void cpu(int argc, char* argv[])
{
    int   x, y, count;
    long double zr, zi, betar, betai, a, b;
    long double rsquared, isquared, mr, mi, msquared;

    double LEFT, BOTTOM, RIGHT, TOP, SIZE, MaxIters;

    if (argc == 1)
    {
        LEFT = -10.;
        BOTTOM = -10.;
        RIGHT = 10.;
        TOP = 10.;
        SIZE = 40;
        MaxIters = 80.;
    }
    else
    {
        LEFT = atof(argv[1]);
        BOTTOM = atof(argv[2]);
        RIGHT = atof(argv[3]);
        TOP = atof(argv[4]);
        SIZE = atof(argv[5]);
        MaxIters = atof(argv[6]);
    }

    char* arr = (char*) malloc(SIZE*SIZE*sizeof(char));
    for (int i = 0; i < SIZE * SIZE; i++)
        arr[i] = ' ';

    for (y = 0; y < SIZE; y++)
    {
        for (x = 0; x < SIZE; x++)
        {
            zr = 1.0;
            zi = 0.0;
            betar = LEFT + x * (RIGHT - LEFT) / SIZE;
            betai = BOTTOM + y * (TOP - BOTTOM) / SIZE;

            a = 0.5 * (1 - betar);
            b = -0.5 * betai;
            rsquared = zr * zr;
            isquared = zi * zi;

            for (count = 0; rsquared + isquared >= 2.25 / (betar * betar + betai * betai) && count < MaxIters; count++)
            {
                mr = 2 * a - 1 + exp(a * zr - b * zi) * cos(a * zi + b * zr);
                mi = 2 * b + exp(a * zr - b * zi) * sin(a * zi + b * zr);
                msquared = mr * mr + mi * mi;
                zr = 1 - 2 * (a * mr + b * mi) / msquared;
                zi = 2 * (a * mi - b * mr) / msquared;
                rsquared = zr * zr;
                isquared = zi * zi;
            }

            if (rsquared + isquared >= 2.25 / (betar * betar + betai * betai))
                arr[x * int(SIZE)+y] = '*';
            else
                arr[x * int(SIZE) + y] = '.';
        }
    }
    for (int i = 0; i < SIZE; i++)
    {
        for (int j = 0; j < SIZE; j++)
            printf("%c", arr[j * int(SIZE) + i]);
        printf("\n");
    }
}

void gpu(int argc, char* argv[])
{
    int   x, y, count;
    long double zr, zi, betar, betai, a, b;
    long double rsquared, isquared, mr, mi, msquared;

    double LEFT, BOTTOM, RIGHT, TOP, SIZE, MaxIters;

    if (argc == 1)
    {
        LEFT = -10.;
        BOTTOM = -10.;
        RIGHT = 10.;
        TOP = 10.;
        SIZE = 40;
        MaxIters = 80.;
    }
    else
    {
        LEFT = atof(argv[1]);
        BOTTOM = atof(argv[2]);
        RIGHT = atof(argv[3]);
        TOP = atof(argv[4]);
        SIZE = atof(argv[5]);
        MaxIters = atof(argv[6]);
    }

    char* arr = (char*)malloc(SIZE * SIZE * sizeof(char));
    for (int i = 0; i < SIZE * SIZE; i++)
        arr[i] = ' ';

    char* dev_a = 0;
  
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, SIZE*SIZE * sizeof(char));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, arr, SIZE*SIZE * sizeof(char), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    dim3 threads = 500;
    dim3 blocks = (SIZE * SIZE) / threads.x + 1;
    addKernel << <blocks, threads>> > (dev_a, LEFT, BOTTOM, RIGHT, TOP, SIZE, MaxIters);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }

    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(arr, dev_a, SIZE*SIZE * sizeof(char), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_a);

    cudaStatus = hipDeviceReset();
 
    for (int i = 0; i < SIZE; i++)
    {
        for (int j = 0; j < SIZE; j++)
            printf("%c", arr[j * int(SIZE) + i]);
        printf("\n");
    }
}

int main(int argc, char* argv[])
{
    //cpu(argc, argv);
    gpu(argc, argv);
    return 0;
}